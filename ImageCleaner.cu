#include "hip/hip_runtime.h"
#include "ImageCleaner.h"
#include <math.h>

#ifndef SIZEX
#error Please define SIZEX.
#endif
#ifndef SIZEY
#error Please define SIZEY.
#endif
#define PI 3.14159265

//----------------------------------------------------------------
// TODO:  CREATE NEW KERNELS HERE.  YOU CAN PLACE YOUR CALLS TO
//        THEM IN THE INDICATED SECTION INSIDE THE 'filterImage'
//        FUNCTION.
//
// BEGIN ADD KERNEL DEFINITIONS
//----------------------------------------------------------------


__global__ void fftx(float *device_real, float *device_imag, int size_x, int size_y)
{
  //draw into shared memory from global
  //__shared__ float realOutBuffer[SIZEX];
  //__shared__ float imagOutBuffer[SIZEX];
  float realOutVal;
  float imagOutVal;
  float threadDeviceReal[size_x * size_y];
  float threadDeviceImag[size_x * size_y];

  __shared__ float fft_real[SIZEY];
  __shared__ float fft_imag[SIZEY];

  for (int n = 0; n < size_y; n++) {
    float term = -2 * PI * threadIdx.x * n / size_y;
    fft_real[n] = cos(term);
    fft_imag[n] = sin(term);
  }

  //realOutBuffer[threadIdx.x] = 0.0f; //we can make this local to each thread--no need for shared!
  //imagOutBuffer[threadIdx.x] = 0.0f;

  realOutVal = 0.0f;
  imagOutVal = 0.0f;
  for (int n = 0; n < size_y; n++) {
    realOutVal += (device_real[blockIdx.x*size_y + n] * fft_real[n]) - (device_imag[blockIdx.x*size_y + n] * fft_imag[n]);
    imagOutVal += (device_imag[blockIdx.x*size_y + n] * fft_real[n]) + (device_real[blockIdx.x*size_y + n] * fft_imag[n]);
  }

  //__syncthreads(); //we don't need to sync because writing to different spots
  // threadDeviceReal[blockIdx.x*size_y + threadIdx.x] = realOutVal;
  // threadDeviceImag[blockIdx.x*size_y + threadIdx.x] = imagOutVal;

  device_real[blockIdx.x*size_y + threadIdx.x] = realOutVal;;
  device_imag[blockIdx.x*size_y + threadIdx.x] = imagOutVal;
}

__global__ void ifftx(float *device_real, float *device_imag, int size_x, int size_y)
{
  //__shared__ float realOutBuffer[SIZEX];
  //__shared__ float imagOutBuffer[SIZEX];
  float realOutVal;
  float imagOutVal;
  // float threadDeviceReal[size_x * size_y];
  // float threadDeviceImag[size_x * size_y];
  __shared__ float fft_real[SIZEY];
  __shared__ float fft_imag[SIZEY];

  for (int n = 0; n < size_y; n++) {
    float term = 2 * PI * threadIdx.x * n / size_y;
    fft_real[n] = cos(term);
    fft_imag[n] = sin(term);
  }

  realOutVal = 0.0f;
  imagOutVal = 0.0f;
  for (int n = 0; n < size_y; n++) {
    realOutVal += (device_real[blockIdx.x*size_y + n] * fft_real[n]) - (device_imag[blockIdx.x*size_y + n] * fft_imag[n]);
    imagOutVal += (device_imag[blockIdx.x*size_y + n] * fft_real[n]) + (device_real[blockIdx.x*size_y + n] * fft_imag[n]);
  }
  
  realOutVal /= size_y;
  imagOutVal /= size_y;

  //__syncthreads();

  // threadDeviceReal[blockIdx.x*size_y + threadIdx.x] = realOutVal;
  // threadDeviceImag[blockIdx.x*size_y + threadIdx.x] = imagOutVal;

  device_real[blockIdx.x*size_y + threadIdx.x] = realOutVal;
  device_imag[blockIdx.x*size_y + threadIdx.x] = imagOutVal;
}

__global__ void ffty(float *device_real, float *device_imag, int size_x, int size_y)
{
  //__shared__ float realOutBuffer[SIZEY];
  //__shared__ float imagOutBuffer[SIZEY];
  float realOutVal;
  float imagOutVal;
  __shared__ float fft_real[SIZEX];
  __shared__ float fft_imag[SIZEX];

  for (int n = 0; n < size_x; n++) {
    float term = -2 * PI * threadIdx.x * n / size_x;
    fft_real[n] = cos(term);
    fft_imag[n] = sin(term);
  }

  realOutVal = 0.0f;
  imagOutVal = 0.0f;
  for (int n = 0; n < size_x; n++) {
    realOutVal += (device_real[n*size_x + blockIdx.x] * fft_real[n]) - (device_imag[n*size_x + blockIdx.x] * fft_imag[n]);
    imagOutVal += (device_imag[n*size_x + blockIdx.x] * fft_real[n]) + (device_real[n*size_x + blockIdx.x] * fft_imag[n]);
  }

  //__syncthreads();
  device_real[threadIdx.x*size_x + blockIdx.x] = realOutVal;
  device_imag[threadIdx.x*size_x + blockIdx.x] = imagOutVal;
}

__global__ void iffty(float *device_real, float *device_imag, int size_x, int size_y)
{
  //__shared__ float realOutBuffer[SIZEY];
  //__shared__ float imagOutBuffer[SIZEY];
  float realOutVal;
  float imagOutVal;
  __shared__ float fft_real[SIZEX];
  __shared__ float fft_imag[SIZEX];

  for (int n = 0; n < size_x; n++) {
    float term = 2 * PI * threadIdx.x * n / size_x;
    fft_real[n] = cos(term);
    fft_imag[n] = sin(term);
  }

  realOutVal = 0.0f;
  imagOutVal = 0.0f;
  for (int n = 0; n < size_x; n++) {
    realOutVal += (device_real[n*size_x + blockIdx.x] * fft_real[n]) - (device_imag[n*size_x + blockIdx.x] * fft_imag[n]);
    imagOutVal += (device_imag[n*size_x + blockIdx.x] * fft_real[n]) + (device_real[n*size_x + blockIdx.x] * fft_imag[n]);
  }

  realOutVal /= size_x;
  imagOutVal /= size_x;

  //__syncthreads();
  device_real[threadIdx.x*size_x + blockIdx.x] = realOutVal;
  device_imag[threadIdx.x*size_x + blockIdx.x] = imagOutVal;
}

__global__ void filter(float *device_real, float *device_imag, int size_x, int size_y)
{
  int eightX = size_x/8;
  int eight7X = size_x - eightX;
  int eightY = size_y/8;
  int eight7Y = size_y - eightY;
  if(!(threadIdx.x < eightX && blockIdx.x < eightY) &&
     !(threadIdx.x < eightX && blockIdx.x >= eight7Y) &&
     !(threadIdx.x >= eight7X && blockIdx.x < eightY) &&
     !(threadIdx.x >= eight7X && blockIdx.x >= eight7Y))
  {
    // Zero out these values
    device_real[threadIdx.x*size_y + blockIdx.x] = 0;
    device_imag[threadIdx.x*size_y + blockIdx.x] = 0;
  }
}



//----------------------------------------------------------------
// END ADD KERNEL DEFINTIONS
//----------------------------------------------------------------

__host__ float filterImage(float *real_image, float *imag_image, int size_x, int size_y)
{
  // check that the sizes match up
  assert(size_x == SIZEX);
  assert(size_y == SIZEY);

  int matSize = size_x * size_y * sizeof(float);

  // These variables are for timing purposes
  float transferDown = 0, transferUp = 0, execution = 0;
  hipEvent_t start,stop;

  CUDA_ERROR_CHECK(hipEventCreate(&start));
  CUDA_ERROR_CHECK(hipEventCreate(&stop));

  // Create a stream and initialize it
  hipStream_t filterStream;
  CUDA_ERROR_CHECK(hipStreamCreate(&filterStream));

  // Alloc space on the device
  float *device_real, *device_imag;
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_real, matSize));
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_imag, matSize));

  // Start timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));
  
  // Here is where we copy matrices down to the device 
  CUDA_ERROR_CHECK(hipMemcpy(device_real,real_image,matSize,hipMemcpyHostToDevice));
  CUDA_ERROR_CHECK(hipMemcpy(device_imag,imag_image,matSize,hipMemcpyHostToDevice));
  
  // Stop timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferDown,start,stop));

  // Start timing for the execution
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  //----------------------------------------------------------------
  // TODO: YOU SHOULD PLACE ALL YOUR KERNEL EXECUTIONS
  //        HERE BETWEEN THE CALLS FOR STARTING AND
  //        FINISHING TIMING FOR THE EXECUTION PHASE
  // BEGIN ADD KERNEL CALLS
  //----------------------------------------------------------------

  // This is an example kernel call, you should feel free to create
  // as many kernel calls as you feel are needed for your program
  // Each of the parameters are as follows:
  //    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
  //    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)
  //    3. Always should be '0' unless you read the CUDA manual and learn about dynamically allocating shared memory
  //    4. Stream to execute kernel on, should always be 'filterStream'
  //
  // Also note that you pass the pointers to the device memory to the kernel call
  fftx <<<size_x,size_y,0,filterStream>>> (device_real,device_imag,size_x,size_y);
  ffty <<<size_x,size_y,0,filterStream>>> (device_real,device_imag,size_x,size_y);
  filter <<<size_x,size_y,0,filterStream>>> (device_real,device_imag,size_x,size_y);
  ifftx <<<size_x,size_y,0,filterStream>>> (device_real,device_imag,size_x,size_y);
  iffty <<<size_x,size_y,0,filterStream>>> (device_real,device_imag,size_x,size_y);


  //---------------------------------------------------------------- 
  // END ADD KERNEL CALLS
  //----------------------------------------------------------------

  // Finish timimg for the execution 
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&execution,start,stop));

  // Start timing for the transfer up
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  // Here is where we copy matrices back from the device 
  CUDA_ERROR_CHECK(hipMemcpy(real_image,device_real,matSize,hipMemcpyDeviceToHost));
  CUDA_ERROR_CHECK(hipMemcpy(imag_image,device_imag,matSize,hipMemcpyDeviceToHost));

  // Finish timing for transfer up
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferUp,start,stop));

  // Synchronize the stream
  CUDA_ERROR_CHECK(hipStreamSynchronize(filterStream));
  // Destroy the stream
  CUDA_ERROR_CHECK(hipStreamDestroy(filterStream));
  // Destroy the events
  CUDA_ERROR_CHECK(hipEventDestroy(start));
  CUDA_ERROR_CHECK(hipEventDestroy(stop));

  // Free the memory
  CUDA_ERROR_CHECK(hipFree(device_real));
  CUDA_ERROR_CHECK(hipFree(device_imag));

  // Dump some usage statistics
  printf("CUDA IMPLEMENTATION STATISTICS:\n");
  printf("  Host to Device Transfer Time: %f ms\n", transferDown);
  printf("  Kernel(s) Execution Time: %f ms\n", execution);
  printf("  Device to Host Transfer Time: %f ms\n", transferUp);
  float totalTime = transferDown + execution + transferUp;
  printf("  Total CUDA Execution Time: %f ms\n\n", totalTime);
  // Return the total time to transfer and execute
  return totalTime;
}

